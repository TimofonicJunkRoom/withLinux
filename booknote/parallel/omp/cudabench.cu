#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include "cudabench.h"

__global__ void
_dcopy_cuda (const double * S, double * D, size_t length)
{
  int tid = blockDim.x * blockIdx.x + threadIdx.x; if (tid < length) D[tid] = S[tid];
}

__global__ void
_dscal_cuda (double * x, const double a, size_t n)
{
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < n) x[tid] = x[tid] * a;
}

void
dcopy_cuda (const double * A, double * B, size_t length)
{
  size_t size = sizeof(double) * length;
  // malloc
  double * d_A = NULL, * d_B = NULL;
  hipMalloc ((void**)&d_A, size);
  hipMalloc ((void**)&d_B, size);
  // transter H -> D
  hipMemcpy (d_A, A, size, hipMemcpyHostToDevice);
  // apply kernel
  int threadsperblock = 256;
  int blockspergrid = (length + threadsperblock - 1)/threadsperblock;
  _dcopy_cuda <<<blockspergrid, threadsperblock>>> (d_A, d_B, length);
  // transter D -> H
  hipMemcpy (B, d_B, size, hipMemcpyDeviceToHost);
  // free
  hipFree (d_A);
  hipFree (d_B);
}

void
dscal_cuda (double * x, const double a, size_t n)
{
  size_t size = sizeof(double) * n;
  // malloc
  double * d_A = NULL;
  hipMalloc ((void**)&d_A, size);
  // transter H -> D
  hipMemcpy (d_A, x, size, hipMemcpyHostToDevice);
  // apply kernel
  int threadsperblock = 256;
  int blockspergrid = (n + threadsperblock - 1)/threadsperblock;
  _dscal_cuda <<<blockspergrid, threadsperblock>>> (d_A, a, n);
  // transter D -> H
  hipMemcpy (x, d_A, size, hipMemcpyDeviceToHost);
  // free
  hipFree (d_A);
}
