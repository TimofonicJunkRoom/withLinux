#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include "cudabench.h"

__global__ void
_dcopy_cuda (const double * S, double * D, size_t length)
{
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < length) D[tid] = S[tid];
}

void
dcopy_cuda (const double * A, double * B, size_t length)
{
  size_t size = sizeof(double) * length;
  // malloc
  double * d_A = NULL, * d_B = NULL;
  hipMalloc ((void**)&d_A, size);
  hipMalloc ((void**)&d_B, size);
  // transter H -> D
  hipMemcpy (d_A, A, size, hipMemcpyHostToDevice);
  // apply kernel
  int threadsperblock = 256;
  int blockspergrid = (length + threadsperblock - 1)/threadsperblock;
  _dcopy_cuda <<<blockspergrid, threadsperblock>>> (d_A, d_B, length);
  // transter D -> H
  hipMemcpy (B, d_B, size, hipMemcpyDeviceToHost);
  // free
  hipFree (d_A);
  hipFree (d_B);
}
