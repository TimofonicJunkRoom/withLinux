#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include "cudabench.h"

__global__ void
_dcopy_cuda (double * S, double * D)
{
  int i = threadIdx.x;
  D[i] = S[i];
}

void
dcopy_cuda (double * A, double * B, size_t length)
{
  size_t size = sizeof(double) * length;
  // malloc
  double * d_A = NULL, * d_B = NULL;
  hipMalloc ((void**)&d_A, size);
  hipMalloc ((void**)&d_B, size);
  // transter H -> D
  hipMemcpy (d_A, A, size, hipMemcpyHostToDevice);
  // apply kernel
  _dcopy_cuda <<<1, length>>> (A, B);
  // transter D -> H
  hipMemcpy (B, d_B, size, hipMemcpyDeviceToHost);
  // free
  hipFree (d_A);
  hipFree (d_B);
}
